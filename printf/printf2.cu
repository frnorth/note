
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void Hello_World_2(int n) {

	int id = threadIdx.x;
	n--;
	printf("Hello World! 22222 %d %d\n", id, n);
	if(n > 0) {
		hipStream_t s;
		hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
		Hello_World_2<<< 1, 1 >>>(n); //, 0, s >>>(n);
		hipStreamDestroy(s);
	}

}
