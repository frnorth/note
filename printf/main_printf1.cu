
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void Hello_World() {

	int id = threadIdx.x;
	printf("Hello World! %d\n", id);

}

int main() {

	int n = 10;
	__global__ void Hello_World();
	__global__ void Hello_World_2(int n);

	Hello_World<<<1, 10>>>();
    hipDeviceSynchronize();

	Hello_World_2<<<1, 1>>>(n);
    hipDeviceSynchronize();

}
